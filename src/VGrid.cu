#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, The Bifrost Authors. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * * Redistributions of source code must retain the above copyright
 *   notice, this list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright
 *   notice, this list of conditions and the following disclaimer in the
 *   documentation and/or other materials provided with the distribution.
 * * Neither the name of The Bifrost Authors nor the names of its
 *   contributors may be used to endorse or promote products derived
 *   from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* 

This module adopts the same work distribution strategy outlined in Romein(2011) to grid 
raw antenna voltages generating a X- and Y- complex uv grid. It replaces the earlier romein gridding
kernel as this module implements a higher level of parallelism with improved performance and hence is an 
upgraded version to the romein_kernel. Note that VGrid is specific to EPIC and cannot be used to grid visibilities
in its current form.

*/
#include <iostream>
#include <bifrost/VGrid.h>
#include "romein_kernels.cuh"

#include "assert.hpp"
#include "trace.hpp"
#include "utils.hpp"
#include "cuda.hpp"
#include "cuda/stream.hpp"
#include "Complex.hpp"

#define MAX_THREADS_PER_BLOCK 128
#define MIN_BLOCKS_PER_MP     4

struct __attribute__((aligned(1))) nibble2 {
    // Yikes!  This is dicey since the packing order is implementation dependent!  
    signed char y:4, x:4;
};

struct __attribute__((aligned(1))) blenib2 {
    // Yikes!  This is dicey since the packing order is implementation dependent!
    signed char x:4, y:4;
};

template<typename RealType>
__host__ __device__
inline Complex<RealType> Complexfcma(Complex<RealType> x, Complex<RealType> y, Complex<RealType> d) {
    RealType real_res;
    RealType imag_res;

    real_res = (x.x *  y.x) + d.x;
    imag_res = (x.x *  y.y) + d.y;

    real_res =  (x.y * y.y) + real_res;
    imag_res = -(x.y * y.x) + imag_res;

    return Complex<RealType>(real_res, imag_res);
}


template<typename InType, typename OutType>
__global__ void 
//__launch_bounds__(MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP)
VGrid_kernel(      int   		       npol,
		   int                         maxsupport, 
		   int                         gridsize, 
		   const int* __restrict__     x,
		   const int* __restrict__     y,
		   const int* __restrict__     z,
		   const OutType* __restrict__ kernels,
		   const InType* __restrict__  d_in,
		   OutType*                    d_out) {
        
	/// Thread and Block Indexes and Dimensions

    int bid_x = blockIdx.x, bid_y = blockIdx.y, bid_z = blockIdx.z ;
    int blk_x = blockDim.x, blk_y = blockDim.y;
    int grid_y = gridDim.y, grid_z = gridDim.z ;
    int illum_x = threadIdx.x, tid_y = threadIdx.y ;

    int vi_s = (bid_y+bid_x*grid_y)*grid_z*blk_y*npol ;
    int grid_s = (bid_y+bid_x*grid_y)*npol*gridsize*gridsize;

    /// Dynamic allocation of shared memory

    extern __shared__ float shared[];
    
    float* xdata = shared;
    float* ydata = xdata + blk_y * npol;
    
    /// Store X- and Y- Locations of Antennas within a thread-block in shared memory

    #pragma unroll
    for(int kk=0;kk<npol;kk++)
    {
          xdata[tid_y*npol + kk] = x[vi_s + npol *(bid_z*blk_y+tid_y)+kk];	
	  ydata[tid_y*npol + kk] = y[vi_s + npol *(bid_z*blk_y+tid_y)+kk];
    }  
    __syncthreads();

       int myU = illum_x% maxsupport; int myV = illum_x / maxsupport;
       float grid_point_u = myU; float grid_point_v = myV;
       OutType sum = OutType(0.0, 0.0);
  
       #pragma unroll
       for(int pol = 0; pol < npol;pol++) 
       {
                 
	    float xl = xdata[tid_y*npol+pol]; 
	    float yl = ydata[tid_y*npol+pol];
            // Determine convolution point. This is basically just an
            // optimised way to calculate.
            float myConvU = 0.0; 
	    float myConvV = 0.0;
            if( maxsupport > 1 ) 
	    {
                myConvU = (int(xl) - myU) % maxsupport; myConvV = (int(yl) - myV) % maxsupport;    
                if (myConvU < 0) myConvU += maxsupport; if (myConvV < 0) myConvV += maxsupport;
            } 
            // Determine grid point. Because of the above we know here that
           float myGridU = xl + myConvU; float myGridV = yl + myConvV;
           
       	   // If grid-point changes for a given illumination pattern 

          if (!(myGridU == grid_point_u && myGridV == grid_point_v)) 
	  { // Atomic add to grid is now removed from this kernel after careful examining and consideration for race-conditions
               if( grid_point_u >= 0 && grid_point_u < gridsize && \
                    grid_point_v >= 0 && grid_point_v < gridsize ) 
	        {
                       d_out[grid_s + pol*gridsize*gridsize + gridsize*int(grid_point_v) + int(grid_point_u)].x+= sum.x;
                       d_out[grid_s + pol*gridsize*gridsize + gridsize*int(grid_point_v) + int(grid_point_u)].y+= sum.y;  
                }
	        // Switch to new point
                sum = OutType(0.0, 0.0);
                grid_point_u = myGridU; grid_point_v = myGridV;
           }
            //TODO: Re-do the w-kernel/gcf for our data.
            OutType px = kernels[((bid_z*blk_y+tid_y)*npol+vi_s+pol)*maxsupport*maxsupport + int(myConvV) * maxsupport + int(myConvU)];
            // Sum up
            InType temp = d_in[(bid_z*blk_y+tid_y)*npol+vi_s+pol];
            OutType vi_v = OutType(temp.x, temp.y);
            sum=Complexfcma(px, vi_v, sum);     
 
            /// Add to the grid within an illumination pattern

            if( grid_point_u >= 0 && grid_point_u < gridsize && \
                grid_point_v >= 0 && grid_point_v < gridsize ) 
	    {
          
                d_out[grid_s + pol*gridsize*gridsize + gridsize*int(grid_point_v) + int(grid_point_u)].x+= sum.x;
                d_out[grid_s + pol*gridsize*gridsize + gridsize*int(grid_point_v) + int(grid_point_u)].y+= sum.y
          }
       }/// End of polarization loop
    __syncthreads();
 }

template<typename InType, typename OutType>
inline void launch_VGrid_kernel(int      nbaseline,
                                 int      npol,
                                 bool     polmajor,
                                 int      maxsupport, 
                                 int      gridsize, 
                                 int      nbatch,
                                 int      nchan,
				 float*     xpos,
                                 float*     ypos,
                                 float*     zpos,
                                 OutType* kernels,
                                 InType*  d_in,
                                 OutType* d_out,
                                 hipStream_t stream=0) {
    
    hipDeviceProp_t dev;
    hipError_t error;
    error = hipGetDeviceProperties(&dev, 0);
    if(error != hipSuccess) printf("Error: %s\n", hipGetErrorString(error));
    int block_x=maxsupport*maxsupport ;
    int block_y ;
    if(block_x==1)block_y=std::min(nbaseline,dev.maxThreadsPerBlock/2);
    else block_y = std::min((int)nbaseline/block_x,dev.maxThreadsPerBlock/2);
    int grid_z=nbaseline/block_y ;
    dim3 block(block_x,block_y);
    if(polmajor)npol=1;
    dim3 grid(nbatch, nchan, grid_z);
    
    void* args[] = {&npol,
                    &maxsupport,
                    &gridsize, 
                    &xpos,
                    &ypos,
                    &zpos,
                    &kernels,
                    &d_in,
                    &d_out};
    size_t loc_size = 2 * block.y * npol * sizeof(float);
    BF_CHECK_CUDA_EXCEPTION(hipLaunchKernel((void*)VGrid_kernel<InType,OutType>,
						 grid, block,&args[0], loc_size, stream),BF_STATUS_INTERNAL_ERROR);
    
}

class BFVGrid_impl {
    typedef int    IType;
    typedef double FType;
public: // HACK WAR for what looks like a bug in the CUDA 7.0 compiler
    typedef float  DType;
private:
    IType        _nbaseline;
    IType        _npol;
    bool         _polmajor;
    IType        _maxsupport;
    IType        _gridsize;
    IType        _nxyz = 0;
    float*         _x = NULL;
    float*         _y = NULL;
    float*         _z = NULL;    
    IType        _nkernels = 0;
    BFdtype      _tkernels = BF_DTYPE_INT_TYPE;
    void*        _kernels = NULL;
    hipStream_t _stream;
public:
    BFVGrid_impl() : _nbaseline(1), _npol(1), _polmajor(true), \
                      _maxsupport(1), _stream(g_cuda_stream) {}
    inline IType nbaseline()  const { return _nbaseline;  }
    inline IType npol()       const { return _npol;       }
    inline bool polmajor()    const { return _polmajor;   }
    inline IType maxsupport() const { return _maxsupport; }
    inline IType gridsize()   const { return _gridsize;   }
    inline IType nxyz()       const { return _nxyz;       }
    inline IType nkernels()   const { return _nkernels;   }
    inline IType tkernels()   const { return _tkernels;   }
    void init(IType nbaseline,
              IType npol,
              bool  polmajor,
              IType maxsupport, 
              IType gridsize) {
        BF_TRACE();
        _nbaseline  = nbaseline;
        _npol       = npol;
        _polmajor   = polmajor;
        _maxsupport = maxsupport;
        _gridsize   = gridsize;
    }
    void set_positions(BFarray const* positions) { 
        BF_TRACE();
        BF_TRACE_STREAM(_stream);
        //BF_ASSERT_EXCEPTION(positions->dtype == BF_DTYPE_I32, BF_STATUS_UNSUPPORTED_DTYPE);
        
        int npositions = positions->shape[1];
        int stride = positions->shape[1];
	for(int i=2; i<positions->ndim-2; ++i) {
            npositions *= positions->shape[i];
	    stride *= positions->shape[i];
	}
	stride *= positions->shape[positions->ndim-2];
	stride *= positions->shape[positions->ndim-1];
	_nxyz = npositions;
        _x = (float *) positions->data;
        _y = _x + stride;
        _z = _y + stride;
    }
    void set_kernels(BFarray const* kernels) {
        BF_TRACE();
        BF_TRACE_STREAM(_stream);
        BF_ASSERT_EXCEPTION((kernels->dtype == BF_DTYPE_CF32) \
                                              || (kernels->dtype == BF_DTYPE_CF64), BF_STATUS_UNSUPPORTED_DTYPE);
        
        int nkernels = kernels->shape[0];
        for(int i=1; i<kernels->ndim-4; ++i) {
            nkernels *= kernels->shape[i];
        }
        
        _nkernels = nkernels;
        _tkernels = kernels->dtype;
        _kernels = (void*) kernels->data;
    }
    void execute(BFarray const* in, BFarray const* out, int nbatch, int nchan) {
        BF_TRACE();
        BF_TRACE_STREAM(_stream);
        BF_ASSERT_EXCEPTION(_x != NULL, BF_STATUS_INVALID_STATE);
	BF_ASSERT_EXCEPTION(_y != NULL, BF_STATUS_INVALID_STATE);
	BF_ASSERT_EXCEPTION(_z != NULL, BF_STATUS_INVALID_STATE);
        BF_ASSERT_EXCEPTION(_kernels != NULL, BF_STATUS_INVALID_STATE);
        BF_ASSERT_EXCEPTION((out->dtype == BF_DTYPE_CF32) \
                                          || (out->dtype == BF_DTYPE_CF64), BF_STATUS_UNSUPPORTED_DTYPE);
        
        BF_CHECK_CUDA_EXCEPTION(hipGetLastError(), BF_STATUS_INTERNAL_ERROR);
        
               
#define LAUNCH_VGRID_KERNEL(IterType,OterType) \
        launch_VGrid_kernel(_nbaseline, _npol, _polmajor, _maxsupport, _gridsize, nbatch, nchan, \
                             _x, _y, _z, (OterType)_kernels,		\
                             (IterType)in->data, (OterType)out->data, \
                             _stream)
        
        switch( in->dtype ) {
            case BF_DTYPE_CI4:
                if( in->big_endian ) {
                    switch( out->dtype ) {
                        case BF_DTYPE_CF32: LAUNCH_VGRID_KERNEL(nibble2*, Complex32*);  break;
                        case BF_DTYPE_CF64: LAUNCH_VGRID_KERNEL(nibble2*, Complex64*);  break;
                        default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                    };
                } else {
                    switch( out->dtype ) {
                        case BF_DTYPE_CF32: LAUNCH_VGRID_KERNEL(blenib2*, Complex32*);  break;
                        case BF_DTYPE_CF64: LAUNCH_VGRID_KERNEL(blenib2*, Complex64*);  break;
                        default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                    };
                }
                break;
            case BF_DTYPE_CI8:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_VGRID_KERNEL(char2*, Complex32*);  break;
                    case BF_DTYPE_CF64: LAUNCH_VGRID_KERNEL(char2*, Complex64*);  break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                };
                break;
            case BF_DTYPE_CI16:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_VGRID_KERNEL(short2*, Complex32*); break;
                    case BF_DTYPE_CF64: LAUNCH_VGRID_KERNEL(short2*, Complex64*); break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                }
                break;
            case BF_DTYPE_CI32:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_VGRID_KERNEL(int2*, Complex32*); break;
                    case BF_DTYPE_CF64: LAUNCH_VGRID_KERNEL(int2*, Complex64*); break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                }
                break;
            case BF_DTYPE_CI64:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_VGRID_KERNEL(long2*, Complex32*); break;
                    case BF_DTYPE_CF64: LAUNCH_VGRID_KERNEL(long2*, Complex64*); break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                }
                break;
            case BF_DTYPE_CF32:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_VGRID_KERNEL(float2*, Complex32*);   break;
                    case BF_DTYPE_CF64: LAUNCH_VGRID_KERNEL(float2*, Complex64*);   break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                }
                break;
            case BF_DTYPE_CF64:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_VGRID_KERNEL(double2*, Complex32*);  break;
                    case BF_DTYPE_CF64: LAUNCH_VGRID_KERNEL(double2*, Complex64*);  break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                }
                break;
            default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
        }
#undef LAUNCH_VGRID_KERNEL
        
        BF_CHECK_CUDA_EXCEPTION(hipGetLastError(), BF_STATUS_INTERNAL_ERROR);
    }
    void set_stream(hipStream_t stream) {
        _stream = stream;
    }
};

BFstatus bfVGridCreate(BFvgrid* plan_ptr) {
    BF_TRACE();
    BF_ASSERT(plan_ptr, BF_STATUS_INVALID_POINTER);
    BF_TRY_RETURN_ELSE(*plan_ptr = new BFVGrid_impl(),
                       *plan_ptr = 0);
}

BFstatus bfVGridInit(BFvgrid       plan,
                      BFarray const* positions,
                      BFarray const* kernels,
                      BFsize         gridsize,
                      BFbool         polmajor) {
    BF_TRACE();
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    BF_ASSERT(positions,                                BF_STATUS_INVALID_POINTER);
    BF_ASSERT(positions->ndim >= 4,                     BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(positions->shape[0] == 3, BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(space_accessible_from(positions->space, BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
    BF_ASSERT(kernels,                                BF_STATUS_INVALID_POINTER);
    BF_ASSERT(kernels->ndim >= 5,                     BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(kernels->shape[kernels->ndim-2] \
              == kernels->shape[kernels->ndim-1],     BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(space_accessible_from(kernels->space, BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
    
    // Discover the dimensions of the positions/kernels.
    int npositions, nbaseline, npol, nkernels, maxsupport;
    npositions = positions->shape[1];
    for(int i=2; i<positions->ndim-2; ++i) {
        npositions *= positions->shape[i];
    }
    if( polmajor ) {
         npol = positions->shape[positions->ndim-2];
         nbaseline = positions->shape[positions->ndim-1];
    } else {
        nbaseline = positions->shape[positions->ndim-2];
        npol = positions->shape[positions->ndim-1];
    }
    nkernels = kernels->shape[0];
    for(int i=1; i<kernels->ndim-4; ++i) {
        nkernels *= kernels->shape[i];
    }
    maxsupport = kernels->shape[kernels->ndim-1];
    
    // Validate
    BF_ASSERT(npositions == nkernels, BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(kernels->shape[kernels->ndim-4] \
              == positions->shape[positions->ndim-2], BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(kernels->shape[kernels->ndim-3] \
              == positions->shape[positions->ndim-1], BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(kernels->shape[kernels->ndim-2] \
              == kernels->shape[kernels->ndim-1], BF_STATUS_INVALID_SHAPE);
    
    BF_TRY(plan->init(nbaseline, npol, polmajor, maxsupport, gridsize));
    BF_TRY(plan->set_positions(positions));
    BF_TRY_RETURN(plan->set_kernels(kernels));
}
BFstatus bfVGridSetStream(BFvgrid    plan,
                           void const* stream) {
    BF_TRACE();
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    BF_ASSERT(stream, BF_STATUS_INVALID_POINTER);
    BF_TRY_RETURN(plan->set_stream(*(hipStream_t*)stream));
}
BFstatus bfVGridSetPositions(BFvgrid       plan,
                              BFarray const* positions) {
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    BF_ASSERT(positions,            BF_STATUS_INVALID_POINTER);
    BF_ASSERT(positions->ndim >= 4, BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(positions->shape[0] == 3,                                     BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(space_accessible_from(positions->space,   BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
    if( plan->polmajor() ) {
        BF_ASSERT(positions->shape[positions->ndim-2] == plan->npol(),      BF_STATUS_INVALID_SHAPE  );
        BF_ASSERT(positions->shape[positions->ndim-1] == plan->nbaseline(), BF_STATUS_INVALID_SHAPE  );
    } else {
        BF_ASSERT(positions->shape[positions->ndim-2] == plan->nbaseline(), BF_STATUS_INVALID_SHAPE  );
        BF_ASSERT(positions->shape[positions->ndim-1] == plan->npol(),      BF_STATUS_INVALID_SHAPE  );
    }
    
    BF_TRY_RETURN(plan->set_positions(positions));
}
BFstatus bfVGridSetKernels(BFvgrid       plan, 
                            BFarray const* kernels) {
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    BF_ASSERT(kernels,            BF_STATUS_INVALID_POINTER);
    BF_ASSERT(kernels->ndim >= 5, BF_STATUS_INVALID_SHAPE  );
    if( plan->polmajor() ) {
        BF_ASSERT(kernels->shape[kernels->ndim-4] == plan->npol(),      BF_STATUS_INVALID_SHAPE  );
        BF_ASSERT(kernels->shape[kernels->ndim-3] == plan->nbaseline(), BF_STATUS_INVALID_SHAPE  );
    } else {
        BF_ASSERT(kernels->shape[kernels->ndim-4] == plan->nbaseline(), BF_STATUS_INVALID_SHAPE  );
        BF_ASSERT(kernels->shape[kernels->ndim-3] == plan->npol(),      BF_STATUS_INVALID_SHAPE  );
    }
    BF_ASSERT(kernels->shape[kernels->ndim-2] == plan->maxsupport(), BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(kernels->shape[kernels->ndim-1] == plan->maxsupport(), BF_STATUS_INVALID_SHAPE  );
    BF_ASSERT(space_accessible_from(kernels->space, BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
    
    BF_TRY_RETURN(plan->set_kernels(kernels));
}
BFstatus bfVGridExecute(BFvgrid          plan,
                         BFarray const* in,
                         BFarray const* out) {
    BF_TRACE();
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    BF_ASSERT(in,   BF_STATUS_INVALID_POINTER);
    BF_ASSERT(out,  BF_STATUS_INVALID_POINTER);
    BF_ASSERT( in->ndim >= 3,          BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(out->ndim == in->ndim+1, BF_STATUS_INVALID_SHAPE);
    
    int nbatch = in->shape[0];
    int nchan = in->shape[1];

    BFarray in_flattened;
    if( in->ndim > 3 ) {
        // Keep the last two dim but attempt to flatten all others
        unsigned long keep_dims_mask = padded_dims_mask(in);
        keep_dims_mask |= 0x1 << (in->ndim-1);
        keep_dims_mask |= 0x1 << (in->ndim-2);
        keep_dims_mask |= 0x1 << (in->ndim-3);
        flatten(in,   &in_flattened, keep_dims_mask);
        in  =  &in_flattened;
        BF_ASSERT(in_flattened.ndim == 3, BF_STATUS_UNSUPPORTED_SHAPE);
    }
  
    BF_ASSERT( in->shape[0] == plan->nxyz(),     BF_STATUS_INVALID_SHAPE);
    BF_ASSERT( in->shape[0] == plan->nkernels(), BF_STATUS_INVALID_SHAPE);
    if( plan->polmajor() ) {
        BF_ASSERT( in->shape[1] == plan->npol(),      BF_STATUS_INVALID_SHAPE);
        BF_ASSERT( in->shape[2] == plan->nbaseline(), BF_STATUS_INVALID_SHAPE);
    } else {
        BF_ASSERT( in->shape[1] == plan->nbaseline(), BF_STATUS_INVALID_SHAPE);
        BF_ASSERT( in->shape[2] == plan->npol(),      BF_STATUS_INVALID_SHAPE);
    }
    
    BFarray out_flattened;
    if( out->ndim > 4 ) {
        // Keep the last three dim but attempt to flatten all others
        unsigned long keep_dims_mask = padded_dims_mask(out);
        keep_dims_mask |= 0x1 << (out->ndim-1);
        keep_dims_mask |= 0x1 << (out->ndim-2);
        keep_dims_mask |= 0x1 << (out->ndim-3);
        keep_dims_mask |= 0x1 << (out->ndim-4);
        flatten(out,   &out_flattened, keep_dims_mask);
        out  =  &out_flattened;
        BF_ASSERT(out_flattened.ndim == 4, BF_STATUS_UNSUPPORTED_SHAPE);
    }

    BF_ASSERT(out->shape[0] == plan->nxyz(),     BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(out->shape[0] == plan->nkernels(), BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(out->shape[1] == plan->npol(),     BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(out->shape[2] == plan->gridsize(), BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(out->shape[3] == plan->gridsize(), BF_STATUS_INVALID_SHAPE);
    
    BF_ASSERT(out->dtype == plan->tkernels(),    BF_STATUS_UNSUPPORTED_DTYPE);
    
    BF_ASSERT(space_accessible_from( in->space, BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
    BF_ASSERT(space_accessible_from(out->space, BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
    BF_TRY_RETURN(plan->execute(in, out, nbatch, nchan));
}

BFstatus bfVGridDestroy(BFvgrid plan) {
    BF_TRACE();
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    delete plan;
    return BF_STATUS_SUCCESS;
}
