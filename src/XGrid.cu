#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, The Bifrost Authors. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * * Redistributions of source code must retain the above copyright
 *   notice, this list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright
 *   notice, this list of conditions and the following disclaimer in the
 *   documentation and/or other materials provided with the distribution.
 * * Neither the name of The Bifrost Authors nor the names of its
 *   contributors may be used to endorse or promote products derived
 *   from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


/* 

This module takes in the two-dimensional voltage patterns of the
X and Y uv-grids to perform element-wise complex multiplication 
producing the full stokes output sequence in the order XX*, YY*, XY* & YX*.
The products are estimated for every time-stamp and frequency channel separately 
which keep accumulating and updated for the specified inegration time in the output.

*/
#include <iostream>
#include "bifrost/XGrid.h"
#include "assert.hpp"
#include "trace.hpp"
#include "utils.hpp"
#include "cuda.hpp"
#include "cuda/stream.hpp"
#include "Complex.hpp"

struct __attribute__((aligned(1))) nibble2 {
    // Yikes!  This is dicey since the packing order is implementation dependent!  
    signed char y:4, x:4;
};

struct __attribute__((aligned(1))) blenib2 {
    // Yikes!  This is dicey since the packing order is implementation dependent!
    signed char x:4, y:4;
};

template<typename In, typename Out>
__global__ void XGrid_kernel(int npol, int gridsize, int nbatch, int nchan,
		     const In* __restrict__  d_in,
                     Out* d_out){

        int bid_x = blockIdx.x, bid_y = blockIdx.y, bid_z = blockIdx.z ;
        int blk_x = blockDim.x ;
        int grid_x = gridDim.x, grid_y = gridDim.y , grid_z = gridDim.z ;
        int tid_x = threadIdx.x ;
	int pol_skip = grid_z*blk_x;
    
        extern  __shared__ Complex<float> shared[] ;
        In* xx = reinterpret_cast<In *>(shared);
        In* yy = xx + blk_x;
 
        int tt = 1;
        if(npol>1) tt=(int)npol/2;

        int bid  = ((bid_x * grid_y + bid_y) * tt  * grid_z  + bid_z) * blk_x ;
	int bid2 = ((bid_x * grid_y + bid_y) * npol * grid_z  + bid_z) * blk_x ;

        #pragma unroll
	for(int i=0;i<npol;i++){

                xx[tid_x] = d_in[bid+i/2*pol_skip+tid_x];
		yy[tid_x] = d_in[bid+i%2*pol_skip+tid_x];
	        
		d_out[bid2+i*pol_skip+tid_x].x += xx[tid_x].x*yy[tid_x].x + xx[tid_x].y*yy[tid_x].y;  
	       	d_out[bid2+i*pol_skip+tid_x].y += xx[tid_x].y*yy[tid_x].x - xx[tid_x].x*yy[tid_x].y;
	}
        __syncthreads();
}

template<typename In, typename Out>
inline void launch_xgrid_kernel(int npol, bool polmajor, int gridsize, int nbatch, int nchan, 
                               In*  d_in,
                               Out* d_out,
                               hipStream_t stream=0) {
   
    hipDeviceProp_t dev;
    hipError_t error;
    int grid_pix = gridsize * gridsize ;
    error = hipGetDeviceProperties(&dev, 0);
    if(error != hipSuccess) printf("Error: %s\n", hipGetErrorString(error));
    int block_x=std::min(grid_pix, dev.maxThreadsPerBlock/2);
    int grid_z=grid_pix/block_x ;
    dim3 block(block_x,1);
    if(polmajor)npol=1;
    dim3 grid(nbatch, nchan, grid_z);

    void* args[] = {&npol,
                    &gridsize, 
                    &nbatch,
		    &nchan,
		    &d_in,
                    &d_out};
    size_t loc_size=2*block.x;
    BF_CHECK_CUDA_EXCEPTION(hipLaunchKernel((void*)XGrid_kernel<In,Out>,
						 grid, block,
						 &args[0], loc_size*sizeof(Complex<float>), stream),BF_STATUS_INTERNAL_ERROR);    
}

class BFxgrid_impl {
    typedef int    IType;
    typedef double FType;
public: // HACK WAR for what looks like a bug in the CUDA 7.0 compiler
    typedef float  DType;
private:
    bool         _polmajor;
    IType        _gridsize;
    hipStream_t _stream;
public:
    BFxgrid_impl() : _polmajor(true), _stream(g_cuda_stream) {}
    inline bool polmajor()    const { return _polmajor;   }
    inline IType gridsize()   const { return _gridsize;   }
    void init(bool  polmajor, IType gridsize) {
        BF_TRACE();
        _polmajor   = polmajor;
        _gridsize   = gridsize;
    }
   void execute(BFarray const* in, BFarray const* out, int nbatch, int nchan, int npol) {
        BF_TRACE();
        BF_TRACE_STREAM(_stream);
        BF_ASSERT_EXCEPTION((out->dtype == BF_DTYPE_CF32) \
                                          || (out->dtype == BF_DTYPE_CF64), BF_STATUS_UNSUPPORTED_DTYPE);
        
        BF_CHECK_CUDA_EXCEPTION(hipGetLastError(), BF_STATUS_INTERNAL_ERROR);
    
        
#define LAUNCH_XGRID_KERNEL(IterType,OterType) \
        launch_xgrid_kernel(npol, _polmajor, _gridsize, nbatch, nchan, \
                             (IterType)in->data, (OterType)out->data, \
                             _stream)
        
        switch( in->dtype ) {
            case BF_DTYPE_CI4:
                if( in->big_endian ) {
                    switch( out->dtype ) {
                        case BF_DTYPE_CF32: LAUNCH_XGRID_KERNEL(nibble2*, Complex32*);  break;
                        case BF_DTYPE_CF64: LAUNCH_XGRID_KERNEL(nibble2*, Complex64*);  break;
                        default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                    };
                } else {
                    switch( out->dtype ) {
                        case BF_DTYPE_CF32: LAUNCH_XGRID_KERNEL(blenib2*, Complex32*);  break;
                        case BF_DTYPE_CF64: LAUNCH_XGRID_KERNEL(blenib2*, Complex64*);  break;
                        default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                    };
                }
                break;
            case BF_DTYPE_CI8:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_XGRID_KERNEL(char2*, Complex32*);  break;
                    case BF_DTYPE_CF64: LAUNCH_XGRID_KERNEL(char2*, Complex64*);  break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                };
                break;
            case BF_DTYPE_CI16:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_XGRID_KERNEL(short2*, Complex32*); break;
                    case BF_DTYPE_CF64: LAUNCH_XGRID_KERNEL(short2*, Complex64*); break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                }
                break;
            case BF_DTYPE_CI32:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_XGRID_KERNEL(int2*, Complex32*); break;
                    case BF_DTYPE_CF64: LAUNCH_XGRID_KERNEL(int2*, Complex64*); break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                }
                break;
            case BF_DTYPE_CI64:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_XGRID_KERNEL(long2*, Complex32*); break;
                    case BF_DTYPE_CF64: LAUNCH_XGRID_KERNEL(long2*, Complex64*); break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                }
                break;
            case BF_DTYPE_CF32:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_XGRID_KERNEL(float2*, Complex32*);   break;
                    case BF_DTYPE_CF64: LAUNCH_XGRID_KERNEL(float2*, Complex64*);   break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                }
                break;
            case BF_DTYPE_CF64:
                switch( out->dtype ) {
                    case BF_DTYPE_CF32: LAUNCH_XGRID_KERNEL(double2*, Complex32*);  break;
                    case BF_DTYPE_CF64: LAUNCH_XGRID_KERNEL(double2*, Complex64*);  break;
                    default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
                }
                break;
            default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
        }
#undef LAUNCH_XGRID_KERNEL
        
        BF_CHECK_CUDA_EXCEPTION(hipGetLastError(), BF_STATUS_INTERNAL_ERROR);
    }
    void set_stream(hipStream_t stream) {
        _stream = stream;
    }
};

BFstatus bfxGridCreate(BFxgrid* plan_ptr) {
    BF_TRACE();
    BF_ASSERT(plan_ptr, BF_STATUS_INVALID_POINTER);
    BF_TRY_RETURN_ELSE(*plan_ptr = new BFxgrid_impl(),
                       *plan_ptr = 0);
}
BFstatus bfxGridInit(BFxgrid       plan,
                      BFsize         gridsize,
                      BFbool         polmajor) {
    BF_TRACE();
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);    
    BF_TRY_RETURN(plan->init(polmajor, gridsize));
}
BFstatus bfxGridSetStream(BFxgrid    plan,
                           void const* stream) {
    BF_TRACE();
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    BF_ASSERT(stream, BF_STATUS_INVALID_POINTER);
    BF_TRY_RETURN(plan->set_stream(*(hipStream_t*)stream));
}
BFstatus bfxGridExecute(BFxgrid          plan,
                         BFarray const* in,
                         BFarray const* out) {
    BF_TRACE();
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    BF_ASSERT(in,   BF_STATUS_INVALID_POINTER);
    BF_ASSERT(out,  BF_STATUS_INVALID_POINTER);
    BF_ASSERT( in->ndim == 6,          BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(out->ndim == in->ndim-1, BF_STATUS_INVALID_SHAPE);
    BFarray in_flattened;
        
    int nbatch = in->shape[1];
    int nchan = in->shape[2];
    int npol = in->shape[3];
     
    if( in->ndim > 5 ) {
        unsigned long keep_dims_mask = padded_dims_mask(out);
        keep_dims_mask |= 0x1 << (in->ndim-1);
        keep_dims_mask |= 0x1 << (in->ndim-2);
        keep_dims_mask |= 0x1 << (in->ndim-3);
        keep_dims_mask |= 0x1 << (in->ndim-4);
        keep_dims_mask |= 0x1 << (in->ndim-5);
        keep_dims_mask |= 0x1 << (in->ndim-6);
	flatten(in,   &in_flattened, keep_dims_mask);
        in  =  &in_flattened;
       BF_ASSERT(in_flattened.ndim == 6, BF_STATUS_UNSUPPORTED_SHAPE); 
    }

    BFarray out_flattened;
    if( out->ndim > 4 ) {
        unsigned long keep_dims_mask = padded_dims_mask(out);
        keep_dims_mask |= 0x1 << (out->ndim-1);
        keep_dims_mask |= 0x1 << (out->ndim-2);
        keep_dims_mask |= 0x1 << (out->ndim-3);
        keep_dims_mask |= 0x1 << (out->ndim-4);
        flatten(out,   &out_flattened, keep_dims_mask);
        out  =  &out_flattened;
        BF_ASSERT(out_flattened.ndim == 4, BF_STATUS_UNSUPPORTED_SHAPE);
    }

    BF_ASSERT(out->shape[2] == plan->gridsize(), BF_STATUS_INVALID_SHAPE);
    BF_ASSERT(out->shape[3] == plan->gridsize(), BF_STATUS_INVALID_SHAPE);    
    BF_ASSERT(space_accessible_from( in->space, BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
    BF_ASSERT(space_accessible_from(out->space, BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
    BF_TRY_RETURN(plan->execute(in, out, nbatch, nchan, npol));
}

BFstatus bfxGridDestroy(BFxgrid plan) {
    BF_TRACE();
    BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
    delete plan;
    return BF_STATUS_SUCCESS;
}
